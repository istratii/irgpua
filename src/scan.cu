#include "hip/hip_runtime.h"

#include "scan.cuh"

#define MAX_BLOCKS 4096

#define X 0
#define A 1
#define P 2

struct _Setup
{
  int next_block_id;
  int global_sums[4096];
  int local_sums[4096];
  cuda::atomic<int, cuda::thread_scope_device> states[4096];
};

static __global__ void _scan(raft::device_span<int> buffer,
                             raft::device_span<_Setup> setup)
{
  __shared__ unsigned int bid;
  __shared__ unsigned int state;
  extern __shared__ int s_buffer[];

  const unsigned int tid = threadIdx.x;
  _Setup* sptr = setup.data();

  // replace blockIdx.x to avoid dead locks
  // first come first served
  if (tid == 0)
    bid = atomicAdd(&sptr->next_block_id, 1);
  __syncthreads();

  const unsigned int id = bid * blockDim.x + tid;
  const int N = buffer.size();

  s_buffer[tid] = id < N ? buffer[id] : 0;
  __syncthreads();

  // compute local sum
  for (int offset = 1; offset < blockDim.x; offset *= 2)
    {
      int val{};
      __syncthreads();
      if (tid >= offset)
        val = s_buffer[tid - offset];
      __syncthreads();
      if (tid >= offset)
        s_buffer[tid] += val;
    }

  if (tid == blockDim.x - 1)
    {
      sptr->local_sums[bid] = s_buffer[tid];
      if (bid == 0)
        sptr->global_sums[bid] = sptr->local_sums[bid];
      sptr->states[bid].store(bid ? A : P, cuda::memory_order_seq_cst);
    }
  __syncthreads();

  for (int ii = bid - 1; ii >= 0; --ii)
    {
      if (tid == 0)
        while ((state = sptr->states[ii].load(cuda::memory_order_seq_cst)) == X)
          ;
      __syncthreads();

      if (state == A)
        {
          s_buffer[tid] += sptr->local_sums[ii];
          __syncthreads();
        }
      else // P
        {
          s_buffer[tid] += sptr->global_sums[ii];
          if (tid == blockDim.x - 1)
            {
              sptr->global_sums[bid] = s_buffer[tid];
              sptr->states[bid].store(P, cuda::memory_order_seq_cst);
            }
          __syncthreads();
          break;
        }
    }

  if (id < N)
    buffer[id] = s_buffer[tid];
}

static __global__ void
_prepare_buffer_for_exclusive_scan(raft::device_span<int> buffer)
{
  if (threadIdx.x == 0)
    buffer[0] = 0;
}

void scan(rmm::device_uvector<int>& buffer, ScanMode mode)
{
  hipStream_t stream = buffer.stream();

  // prepare setup
  rmm::device_buffer raw_setup(sizeof(_Setup), stream);
  CUDA_CHECK_ERROR(
    hipMemsetAsync(raw_setup.data(), 0, raw_setup.size(), stream));
  _Setup* setup = static_cast<_Setup*>(raw_setup.data());

  CUDA_CHECK_ERROR(hipStreamSynchronize(stream));

  raft::device_span<int> buffer_span(buffer.data(), buffer.size());
  raft::device_span<_Setup> setup_span(setup, 1);

  CUDA_CHECK_ERROR(hipStreamSynchronize(stream));

#define THREADS_PER_BLOCK 1024

  if (mode == SCAN_EXCLUSIVE)
    _prepare_buffer_for_exclusive_scan<<<1, 1, 0, stream>>>(buffer_span);

  _scan<<<(buffer.size() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK,
          THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(int), stream>>>(
    buffer_span, setup_span);

  CUDA_CHECK_ERROR(hipStreamSynchronize(stream));

#undef THREADS_PER_BLOCK
}
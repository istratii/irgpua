#include "hip/hip_runtime.h"


#include "map_fix.cuh"

#ifdef _IRGPUA_GPU
static __global__ void _map_fix(raft::device_span<int> buffer_dspan)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < buffer_dspan.size())
    buffer_dspan[idx] += ASSOC_VAL(idx);
}
#else
struct MapFixFunctor
{
  __device__ int operator()(int idx) const { return idx + ASSOC_VAL(idx); }
};
#endif

void map_fix(raft::device_span<int> buffer_dspan, hipStream_t stream)
{
  raft::common::nvtx::range fscope("map fix");

#ifdef _IRGPUA_GPU
  constexpr unsigned int block_size = 1024;
  const unsigned int grid_size =
    (buffer_dspan.size() + block_size - 1) / block_size;
  _map_fix<<<grid_size, block_size, 0, stream>>>(buffer_dspan);
#else // _IRGPUA_GPU_INDUS
  // thrust::device_ptr<int> buffer_ptr(buffer_dspan.data());
  // thrust::transform(thrust::cuda::par.on(stream), buffer_ptr,
  //                   buffer_ptr + buffer_dspan.size(), buffer_ptr,
  //                   MapFixFunctor());
#endif
}
